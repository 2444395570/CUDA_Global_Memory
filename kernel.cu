#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <memory>

#define N 5


/*
�ô�����ʾ����δ��豸�����н���ȫ���ڴ��д�룬�Լ���δ�������������hipMalloc���з��䣬��ν�ָ��
�ö�ȫ���ڴ��ָ����Ϊ�ں˺������ں˺����ò�ͬ���߳�ID��ֵ������ȫ���ڴ档Ȼ����hipMemcpy�����Ƶ��ڴ��Ա�
��ʾ���ݡ�����ʹ�õ���ȫ���ڴ棬����������������
*/
__global__ void gpu_global_memory(int* d_a) {
	d_a[threadIdx.x] = threadIdx.x;
}

int main(void) {
	int h_a[N];
	int* d_a;
	hipMalloc((void**)&d_a, sizeof(int) * N);
	hipMemcpy((void*)d_a, (void*)h_a, sizeof(int) * N, hipMemcpyHostToDevice);
	gpu_global_memory << <1, N >> > (d_a);
	hipMemcpy((void*)h_a, (void*)d_a, sizeof(int) * N, hipMemcpyDeviceToHost);
	printf("Array in Global Memory is:\n");
	for (int i = 0; i < N; i++)
	{
		printf("At Index:%d -->%d\n", i, h_a[i]);
	}
	return 0;
}